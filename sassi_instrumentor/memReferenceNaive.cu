#include "hip/hip_runtime.h"
#include <cupti.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <sassi/sassi-core.hpp>
#include <sassi/sassi-memory.hpp>
#include "sassi_lazyallocator.hpp"

#define WARP_SIZE   32
#define BUFFER_SIZE (WARP_SIZE*WARP_SIZE*WARP_SIZE)

__managed__ intptr_t sassiReferences[BUFFER_SIZE];
__device__ unsigned int memIndex = 0;

static void sassi_finalize(sassi::lazy_allocator::device_reset_reason reason) {
    FILE *file = fopen("sassi-memReferences.txt", "a");
    fprintf(file, "Memory References:\n");

    for (unsigned i = 0; i <= BUFFER_SIZE; i++) {
        fprintf(file, "%p\n", (void*) sassiReferences[i]);
    }
    fprintf(file, "\n");
    fclose(file);
}

static sassi::lazy_allocator referencesInitializer(
    []() {
        //initialize necessary data structures 
        bzero(sassiReferences, sizeof(sassiReferences));
    }, 
    // get the results after kernel execution
    sassi_finalize);


__device__ void sassi_before_handler(SASSIBeforeParams *bp, SASSIMemoryParams *mp) {
    
    if (bp->GetInstrWillExecute()) {
        //only execute if memory operation is a read or write, to be safe
        if (bp->IsMemRead() || bp->IsMemWrite()) { 
            intptr_t mpAddr = mp->GetAddress();
            intptr_t baseAddr = mpAddr & ~0x1FF; // mask the lower 9 bits off 
            unsigned int currentIndex  = atomicAdd(&memIndex, 1);
            sassiReferences[currentIndex] = baseAddr;
        }
    }
}
