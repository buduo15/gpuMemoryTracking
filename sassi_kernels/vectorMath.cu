#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixAdd(int *A, int *B, int *result, int *length) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < *length) {
        result[threadId] = A[threadId] + B[threadId];
    }
}

void runMatrixAdd() {
    srand(0);
    int maxLength = 10000;
    int length = (rand() % maxLength) + 100; // make sure the length is at least 100
    
    // create arrays for algorithm
    int *A = (int*) malloc (sizeof(int) * length);
    int *B = (int*) malloc(sizeof(int) * length);
    int *results = (int*) malloc(sizeof(int) * length);

    // fill arrays with random numbers
    for (int i = 0; i < length; i++) {
        A[i] = rand() % 1000;
        B[i] = rand() % 1000;
    }

    int *d_A;

    if (hipMalloc((void**) &d_A, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for matrix A on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_A, A, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying matrix A onto gpu");
        exit(-1);
    }
    
    int *d_B;
    if (hipMalloc((void**) &d_B, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for matrix B on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_B, B, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying matrix B onto gpu");
        exit(-1);
    }

    int *d_results;

    if (hipMalloc((void**) &d_results, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for results matrix on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_results, results, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying results matrix onto gpu");
        exit(-1);
    }

    int *d_length;

    if (hipMalloc((void**) &d_length, sizeof(int)) != hipSuccess) {
        printf("Error allocating for length variable on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying length variable onto gpu");
        exit(-1);
    }
    
    int numThreadsPerBlock = min(1024, length); // can have up to 1024 threads per block on our gpu
    int numBlocks = (length + numThreadsPerBlock - 1) / numThreadsPerBlock;

    // TODO: choose more appropriate blocks and threads
    matrixAdd<<<numBlocks,numThreadsPerBlock>>>(d_A, d_B, d_results, d_length);
    hipDeviceSynchronize();
    
    std::cout << "The last error was: ";
    std::cout << hipGetLastError() << std::endl;    

    // get results back
    if (hipMemcpy(results, d_results, length * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("error getting results back from device");
        exit(-1);
    }

    // print results
    for (int i = 0; i < length; i++) {
        printf("%d", results[i]);
        if (i+1 != length) {
            printf(", ");
        }
    }
}

int main (int argc, char** argv) {
    runMatrixAdd();
    return 0;
}

