#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(int *A, int *B, int *result, int *length) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < *length) {
        result[threadId] = A[threadId] + B[threadId];
    }
}

void runMatrixAdd() {
    srand(0);
    int length = rand() + 100; // make sure the length is at least 100
    
    // create arrays for algorithm
    int A[length];
    int B[length];
    int results[length];

    // fill arrays with random numbers
    for (int i = 0; i < length; i++) {
        A[i] = rand();
        B[i] = rand();
    }

    int *d_A;
    
    if (hipMalloc((void**) &d_A, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for matrix A on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_A, A, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying matrix A onto gpu");
        exit(-1);
    }
    
    int *d_B;
    if (hipMalloc((void**) &d_B, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for matrix B on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_B, B, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying matrix B onto gpu");
        exit(-1);
    }

    int *d_results;

    if (hipMalloc((void**) &d_results, length * sizeof(int)) != hipSuccess) {
        printf("Error allocating for results matrix on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_results, results, length * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying results matrix onto gpu");
        exit(-1);
    }

    int *d_length;

    if (hipMalloc((void**) &d_length, sizeof(int)) != hipSuccess) {
        printf("Error allocating for length variable on gpu");
        exit(-1);
    }

    if (hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying length variable onto gpu");
        exit(-1);
    }

    // TODO: choose more appropriate blocks and threads
    matrixAdd<<<100,100>>>(d_A, d_B, d_results, d_length);
    hipDeviceSynchronize();
    
    // get results back
    if (hipMemcpy(results, d_results, length * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("error getting results back from device");
        exit(-1);
    }

    // print results
    for (int i = 0; i < length; i++) {
        printf("%d", results[i]);
        if (i+1 != length) {
            printf(", ");
        }
    }
}


int main(int argc, char** argv) {
    runMatrixAdd();
    return 0;
}
